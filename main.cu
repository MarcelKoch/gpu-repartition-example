#include <iostream>
#include <memory>
#include <vector>

#include <mpi.h>

#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

int main(int argc, char** argv) {

    MPI_Init(&argc, &argv);

    auto comm = MPI_COMM_WORLD;
    int rank;
    int size;

    MPI_Comm_size(comm, &size);
    MPI_Comm_rank(comm, &rank);

    thrust::host_vector<double> host_local_buffer(10, static_cast<double>(rank));
    thrust::host_vector<double> host_shared_buffer(rank == 0 ? size * host_local_buffer.size() : 0, static_cast<double>(-1));

    thrust::device_vector<double> local_buffer = host_local_buffer;
    thrust::device_vector<double> shared_buffer = host_shared_buffer;

    double* dummy;
    hipMalloc(&dummy, 1);

    MPI_Win win;
    MPI_Win_create(dummy, 0, sizeof(double),
        MPI_INFO_NULL, comm, &win);

    MPI_Win_fence(0, win);
    //MPI_Put(local_buffer.data().get(), local_buffer.size(), MPI_DOUBLE, 0,
//        rank * local_buffer.size(), local_buffer.size(),  MPI_DOUBLE, win);
    MPI_Win_fence(0, win);

    host_shared_buffer = shared_buffer;
    if(rank == 0) {
        std::cout << host_shared_buffer.size() << std::endl;
        for (auto i: shared_buffer) {
            std::cout << i << " ";
        }
        std::cout << std::endl;
    }
    MPI_Barrier(comm);

    std::cout << host_shared_buffer.size() << std::endl;

    hipFree(dummy);

    MPI_Finalize();
}
