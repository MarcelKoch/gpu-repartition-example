#include <iostream>
#include <memory>
#include <vector>

#include <mpi.h>

#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

int main(int argc, char** argv) {

    MPI_Init(&argc, &argv);

    auto comm = MPI_COMM_WORLD;
    int rank;
    int size;

    MPI_Comm_size(comm, &size);
    MPI_Comm_rank(comm, &rank);

    thrust::host_vector<double> host_local_buffer(10, static_cast<double>(rank));
    thrust::host_vector<double> host_shared_buffer(rank == 0 ? size * host_local_buffer.size() : 0, static_cast<double>(-1));

    thrust::device_vector<double> local_buffer = host_local_buffer;
    thrust::device_vector<double> shared_buffer = host_shared_buffer;

    MPI_Gather(local_buffer.data().get(), local_buffer.size(), MPI_DOUBLE,
               shared_buffer.data().get(), local_buffer.size(), MPI_DOUBLE, 0, comm);

    host_shared_buffer = shared_buffer;
    if(rank == 0) {
        std::cout << host_shared_buffer.size() << std::endl;
        for (auto i: shared_buffer) {
            std::cout << i << " ";
        }
        std::cout << std::endl;
    }
    MPI_Barrier(comm);

    std::cout << host_shared_buffer.size() << std::endl;

    MPI_Finalize();
}
